#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <ctime>

//Cuda header
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//OpenCV
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/ml/ml.hpp"

//Project
#include "filecon.h"
#include "delaycalc.h"
#include "gpu.cuh"

#define FREQ_FPGA_CLOCK 100000000 //100Mhz
#define FREQ_SAMPLING	40000000  //40Mhz
#define PITCH			0.0005	  //0.5mn
#define SOUND_SPEED		1540	  //1540m/s
#define SIGNAL_SIZE		8192	  //H
#define CHANNEL			32		  //Point
#define SCAN_LINE		81	      //W
#define NBEFOREPULSE	538
#define NRX				32
#define NTX				32
#define HALFN			4097	  //8192/2 +1 for hilbert transform
#define STARTLOG		1e-9	  //Matlab logcompressdb for minimum
#define NUMCOFFILTER	38		  //Cof Filter

using namespace std;
using namespace cv;

int Div0Up(int a, int b)//fix int/int=0
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void delaysum_beamforming(double *output, const double *tdr, const double *raw_signal)
{
	double sum = 0;
	int Ntdr = 0;
	for (int nl = 0; nl < SCAN_LINE; nl++) //81 scanline
	{
		for (int p = 0; p < SIGNAL_SIZE; p++)//1 scanline 8192 point
		{
			sum = 0;
			for (int i = 0; i < CHANNEL; i++)//1 point = 32 channel
			{
				Ntdr = int(tdr[i + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)]);
				if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
				{
					sum += raw_signal[Ntdr + (i * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
				}
			}
			output[p + (nl * SIGNAL_SIZE)] = sum;
		}
	}
}

int main()
{
	//Cpu mem init
	int    dataLength = SIGNAL_SIZE * SCAN_LINE; // 663552
	int	   Fullsize = dataLength * CHANNEL *sizeof(double);
	int    Imgsize = dataLength * sizeof(double);
	int	   *tdfindex = new int[CHANNEL * SIGNAL_SIZE];
	double *t0 = new double[SCAN_LINE];
	double *max_ps_delay = new double[SCAN_LINE];
	double *tdmin = new double[SIGNAL_SIZE];
	double *elementRxs = new double[CHANNEL * SCAN_LINE];
	double *tdf = new double[2 * CHANNEL * SIGNAL_SIZE];
	double *tdds = new double[2 * CHANNEL * SIGNAL_SIZE];
	double *vout = new double[SIGNAL_SIZE * SCAN_LINE];
	double *raw_data = new double[SIGNAL_SIZE * CHANNEL * SCAN_LINE];
	double *tdr = new double[SIGNAL_SIZE * CHANNEL * SCAN_LINE];
	double *filter = new double[SIGNAL_SIZE];
	float2 *vout_com = new float2[SIGNAL_SIZE * SCAN_LINE];
	float2 *filter_com = new float2[SIGNAL_SIZE];

	//Cuda mem init
	float2 *d_Signalfilter;
	float2 *d_filter_com;
	float2 *d_vout;
	double *d_tdr;
	double *d_raw_signal;
	double *d_max;
	double *d_env;
	int *d_mutex;

	loadRawData("D:\\20160613_122303_RawDataD_0.data", raw_data); // channel*scanline size
	loadData("D:\\ultrasound\\loadPsDelay.dat", SCAN_LINE, max_ps_delay);
	loadData("D:\\ultrasound\\Filter\\BandFilter38Cof.dat", NUMCOFFILTER, filter);
	loadElementRxs("D:\\ultrasound\\loadElementRxs.dat", elementRxs); // channel*scanline size

	for (int i = 0; i < SCAN_LINE; i++)
		t0[i] = NBEFOREPULSE + (max_ps_delay[i] / FREQ_FPGA_CLOCK * FREQ_SAMPLING);

	//Filter zero add
	for (int i = 0; i < NUMCOFFILTER; ++i)
	{
		filter_com[i].x = filter[i]; filter_com[i].y = 0;
	}
	for (int i = NUMCOFFILTER; i < SIGNAL_SIZE; ++i)
	{
		filter_com[i].x = 0; filter_com[i].y = 0;
	}

	//calc Delay time
	calc_TimeDelay(tdf, tdmin, NTX * 2, PITCH, SOUND_SPEED, FREQ_SAMPLING); // TDF
	calc_tdds(tdds, NRX * 2, tdf, tdmin, FREQ_SAMPLING); //TDDS	
	calc_tdfindex(tdfindex, NRX, elementRxs);// Index TDF
	calc_tdr(tdr, NRX, tdds, tdfindex, t0);//TDR

	clock_t startTime1 = clock();
	delaysum_beamforming(vout, tdr, raw_data);
	cout << "CPU delaysum_beamforming times = "<<double(clock() - startTime1) / (double)CLOCKS_PER_SEC*1000 << " ms." << endl;

	//CUDA INIT
	hipfftHandle plan;
	hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, 81);
	hipfftHandle plan1;
	hipfftPlan1d(&plan1, SIGNAL_SIZE, HIPFFT_C2C,1);
	hipMalloc((void **)&d_filter_com, SIGNAL_SIZE * sizeof(float2));
	hipMalloc((void **)&d_vout, SIGNAL_SIZE * SCAN_LINE * sizeof(float2));
	hipMalloc((void **)&d_Signalfilter, SIGNAL_SIZE * SCAN_LINE * sizeof(float2));
	hipMalloc((void **)&d_tdr, Fullsize);
	hipMalloc((void **)&d_raw_signal, Fullsize);
	hipMalloc((void **)&d_env, Imgsize);
	hipMalloc((void**)&d_max, sizeof(double));
	hipMalloc((void**)&d_mutex, sizeof(int));
	hipMemset(d_max, 0, sizeof(float));
	hipMemset(d_mutex, 0, sizeof(float));
	hipMemcpy(d_filter_com, filter_com, SIGNAL_SIZE * sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(d_tdr, tdr, Fullsize, hipMemcpyHostToDevice);
	hipMemcpy(d_raw_signal, raw_data, Fullsize, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float mi = 0, sm = 0 ,mh=0 ,mf=0 ,ml=0;

	/*
	for (int nl = 0; nl < SCAN_LINE; nl++){
		hipEventRecord(start);
		beamforming1scanline << < 32, 256 >> >(nl, d_vout, d_tdr, d_raw_signal);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&mi, start, stop);
		sm += mi;
	}
	cout << "Gpu basic Beamfrom times = " << sm << "ms\n";
	*/

	hipEventRecord(start);
	improve<< <dim3(256,1,1),dim3(32,32,1) >> >(d_vout, d_tdr, d_raw_signal);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mi, start, stop);
	cout <<"Gpu Improve beamfrom times = "<<mi<< "ms\n";
	
	//DATAF Output
	/*
	hipMemcpy(vout_com, d_vout, SIGNAL_SIZE * SCAN_LINE * sizeof(float2), hipMemcpyDeviceToHost);
	for (int i = 0; i < 8192 * 81; i++)
		vout[i] = vout_com[i].x;
	writeFile("D:\\ultrasound\\save.dat", dataLength, vout); //output Vout
	*/

	//OLD FILTER SLOW
	/*
	hipfftExecC2C(plan, (hipfftComplex *)d_filter_com, (hipfftComplex *)d_filter_com, HIPFFT_FORWARD);
	for (int nl = 0; nl < 81; nl++ )
	{
		hipfftExecC2C(plan1, (hipfftComplex *)(d_vout + nl*SIGNAL_SIZE), (hipfftComplex *)(d_vout + nl*SIGNAL_SIZE), HIPFFT_FORWARD);
		FilterCalc << <dim3(256, 1, 1), dim3(1024, 1, 1) >> >(d_Signalfilter , d_vout, d_filter_com, nl);
		hilbert_1line_step2 << <dim3(256, 1, 1), dim3(1024, 1, 1) >> >(d_Signalfilter + nl*SIGNAL_SIZE);
		hipfftExecC2C(plan1, (hipfftComplex *)(d_Signalfilter + nl*SIGNAL_SIZE), (hipfftComplex *)(d_Signalfilter + nl*SIGNAL_SIZE), HIPFFT_BACKWARD);
	}
	*/

	//Filter,Hilbert,abs
	hipEventRecord(start);
	hipfftExecC2C(plan, (hipfftComplex *)d_vout, (hipfftComplex *)d_vout, HIPFFT_FORWARD);
	hipfftExecC2C(plan1, (hipfftComplex *)d_filter_com, (hipfftComplex *)d_filter_com, HIPFFT_FORWARD);
	FilterCalcImprove1 << <dim3(256, 1, 1), dim3(32, 32, 1) >> >(d_Signalfilter, d_vout, d_filter_com);
	hilbert_step2 << <dim3(256, 1, 1), dim3(32, 32, 1) >> >(d_Signalfilter);
	hipfftExecC2C(plan, (hipfftComplex *)d_Signalfilter, (hipfftComplex *)d_Signalfilter, HIPFFT_BACKWARD);
	abscomplex << <dim3(256, 1, 1), dim3(32, 32, 1) >> >(d_env, d_Signalfilter);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mh, start, stop);
	cout << "Gpu Filter,Hilbert,abs times = " << mh << "ms\n";
	
	//Median Filter
	hipEventRecord(start);
	Gpu_median_filter << <dim3(780, 1, 1), dim3(8, 128, 1) >> >(d_env, d_env, SIGNAL_SIZE, SCAN_LINE); // (x/FREQ_SAMPLING*SOUND_SPEED/2*100) = cm , if 12 cm x=6234 ,6234/8 = 780 Fullpic948
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mf, start, stop);
	cout << "Gpu Median Filter times = " << mf << "ms\n";
	
	//LogCompression
	hipEventRecord(start);
	find_maximum << < 32, 256 >> >(d_env, d_max, d_mutex, SIGNAL_SIZE*SCAN_LINE); //<-danger
	logCompressDB << <dim3(256, 1, 1), dim3(32, 32, 1) >> >(d_env, d_max);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ml, start, stop);
	cout << "Gpu LogCompression times = " << ml << "ms\n";

	//Output
	hipMemcpy(vout, d_env, Imgsize, hipMemcpyDeviceToHost);
	writeFile("D:\\ultrasound\\save.dat", dataLength, vout); //output Vout
	
	for (int i = 0; i < 8192 * 81; i++)
		vout[i] = (vout[i] + 180) / 255;
	Mat A = Mat(81, 8192, CV_64FC1, vout);
	A = A(Rect(0, 0, 6234, 81)); //Crop 6234 = 12  Cm
	resize(A, A, Size(768, 243), CV_INTER_CUBIC);
	transpose(A, A);
	imshow("Image", A);
	waitKey(0);

	delete tdfindex;
	delete raw_data;
	delete max_ps_delay;
	delete elementRxs;
	delete t0;
	delete tdf;
	delete tdds;
	delete tdmin;
	delete tdr;
	delete vout;
	delete filter;
	delete vout_com;
	delete filter_com;

	hipfftDestroy(plan);
	hipfftDestroy(plan1);
	hipFree(d_vout);
	hipFree(d_tdr);
	hipFree(d_raw_signal);
	hipFree(d_mutex);
	hipFree(d_max);
	hipFree(d_env);
	hipFree(d_Signalfilter);
}
