#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <ctime>

//Cuda header
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//OpenCV
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/ml/ml.hpp"

//Project
#include "filecon.h"
#include "delaycalc.h"
#include "gpu.cuh"

#define FREQ_FPGA_CLOCK 100000000 //100Mhz
#define FREQ_SAMPLING	40000000  //40Mhz
#define PITCH			0.0005	  //0.5mn
#define SOUND_SPEED		1540	  //1540m/s
#define SIGNAL_SIZE		8192	  //H
#define CHANNEL			32		  //Point
#define SCAN_LINE		81	      //W
#define NBEFOREPULSE	538
#define NRX				32
#define NTX				32
#define HALFN			4097	  //8192/2 +1 for hilbert transform
#define EPSILON			1e-6	  //Matlab logcompressdb for minimum

using namespace std;
using namespace cv;

__global__ void FilterCalc(float2 *signal, float2 *filter)
{
	const int nThdx = blockDim.x * gridDim.x;
	const int tIDx = blockIdx.x * blockDim.x + threadIdx.x;
	for (int p = tIDx; p < SIGNAL_SIZE; p += nThdx)//1 scanline 8192 point
	{
		signal[p].x = signal[p].x * filter[p].x - signal[p].y * filter[p].y;
		signal[p].y = signal[p].x * filter[p].y + signal[p].y * filter[p].x;
	}
}

int Div0Up(int a, int b)//fix int/int=0
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}


void delaysum_beamforming(double *output, const double *tdr, const double *raw_signal)
{
	double sum = 0;
	int Ntdr = 0;
	for (int nl = 0; nl < SCAN_LINE; nl++) //81 scanline
	{
		for (int p = 0; p < SIGNAL_SIZE; p++)//1 scanline 8192 point
		{
			sum = 0;
			for (int i = 0; i < CHANNEL; i++)//1 point = 32 channel
			{
				Ntdr = int(tdr[i + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)]);
				if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
				{
					sum += raw_signal[Ntdr + (i * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
				}
			}
			output[p + (nl * SIGNAL_SIZE)] = sum;
		}
	}
}

int main()
{
	//Cpu mem init
	int    dataLength = SIGNAL_SIZE * SCAN_LINE; // 663552
	int	   Fullsize = dataLength * CHANNEL *sizeof(double);
	int    Imgsize = dataLength * sizeof(double);
	int	   *tdfindex = new int[CHANNEL * SIGNAL_SIZE];
	double *t0 = new double[SCAN_LINE];
	double *max_ps_delay = new double[SCAN_LINE];
	double *tdmin = new double[SIGNAL_SIZE];
	double *elementRxs = new double[CHANNEL * SCAN_LINE];
	double *tdf = new double[2 * CHANNEL * SIGNAL_SIZE];
	double *tdds = new double[2 * CHANNEL * SIGNAL_SIZE];
	double *vout = new double[SIGNAL_SIZE * SCAN_LINE];
	double *raw_data = new double[SIGNAL_SIZE * CHANNEL * SCAN_LINE];
	double *tdr = new double[SIGNAL_SIZE * CHANNEL * SCAN_LINE];
	double *filter = new double[SIGNAL_SIZE];
	float2 *vout_com = new float2[SIGNAL_SIZE * SCAN_LINE];
	float2 *filter_com = new float2[SIGNAL_SIZE];

	//Cuda mem init
	float2 *d_filter_com;
	float2 *d_vout;
	double *d_tdr;
	double *d_raw_signal;
	double *d_max;
	double *d_env;
	int *d_mutex;

	loadRawData("D:\\loadData.dat", raw_data); // channel*scanline size
	loadData("D:\\ultrasound\\loadPsDelay.dat", SCAN_LINE, max_ps_delay);
	loadData("D:\\ultrasound\\loadFilter.dat", 11, filter);
	loadElementRxs("D:\\ultrasound\\loadElementRxs.dat", elementRxs); // channel*scanline size

	for (int i = 0; i < SCAN_LINE; i++)
		t0[i] = NBEFOREPULSE + (max_ps_delay[i] / FREQ_FPGA_CLOCK * FREQ_SAMPLING);

	for (int i = 0; i < 11; ++i)
	{ 
		filter_com[i].x = filter[i] ; filter_com[i].y = 0;
	}
	for (int i = 11; i < SIGNAL_SIZE; ++i)
	{
		filter_com[i].x = 0; filter_com[i].y = 0;
	}

	calc_TimeDelay(tdf, tdmin, NTX * 2, PITCH, SOUND_SPEED, FREQ_SAMPLING); // TDF
	calc_tdds(tdds, NRX * 2, tdf, tdmin, FREQ_SAMPLING); //TDDS	
	calc_tdfindex(tdfindex, NRX, elementRxs);// Index TDF
	calc_tdr(tdr, NRX, tdds, tdfindex, t0);//TDR

	clock_t startTime1 = clock();
	delaysum_beamforming(vout, tdr, raw_data);
	cout << "CPU delaysum_beamforming times = "<<double(clock() - startTime1) / (double)CLOCKS_PER_SEC*1000 << " ms." << endl;

	hipfftHandle plan;
	hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, SCAN_LINE);
	hipfftHandle plan1;
	hipfftPlan1d(&plan1, SIGNAL_SIZE, HIPFFT_C2C, 1);

	hipMalloc((void **)&d_filter_com, SIGNAL_SIZE * sizeof(float2));
	hipMalloc((void **)&d_vout, SIGNAL_SIZE * SCAN_LINE * sizeof(float2));
	hipMalloc((void **)&d_tdr, Fullsize);
	hipMalloc((void **)&d_raw_signal, Fullsize);
	hipMalloc((void **)&d_env, Imgsize);
	hipMalloc((void**)&d_max, sizeof(double));
	hipMalloc((void**)&d_mutex, sizeof(int));

	hipMemset(d_max, 0, sizeof(float));
	hipMemset(d_mutex, 0, sizeof(float));

	hipMemcpy(d_filter_com, filter_com, SIGNAL_SIZE * sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(d_tdr, tdr, Fullsize, hipMemcpyHostToDevice);
	hipMemcpy(d_raw_signal, raw_data, Fullsize, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float mi = 0, sm = 0 ,mh=0 ,mf=0 ,ml=0;

	/*
	for (int nl = 0; nl < SCAN_LINE; nl++){
		hipEventRecord(start);
		beamforming1scanline << < 32, 256 >> >(nl, d_vout, d_tdr, d_raw_signal);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&mi, start, stop);
		sm += mi;
	}
	cout << "Gpu basic Beamfrom times = " << sm << "ms\n";
	*/

	hipEventRecord(start);
	improve<< <dim3(256,1,1),dim3(32,32,1) >> >(d_vout, d_tdr, d_raw_signal);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mi, start, stop);
	cout <<"Gpu Improve beamfrom times = "<<mi<< "ms\n";
	
	hipEventRecord(start);
	hipfftExecC2C(plan, (hipfftComplex *)d_vout, (hipfftComplex *)d_vout, HIPFFT_FORWARD);
	hipfftExecC2C(plan1, (hipfftComplex *)d_filter_com, (hipfftComplex *)d_filter_com, HIPFFT_FORWARD);
	FilterCalc << <dim3(8, 1, 1), dim3(1024, 1, 1) >> >(d_vout, d_filter_com);
	hilbert_step2 << <dim3(256, 1, 1), dim3(32, 32, 1) >> >(d_vout);
	hipfftExecC2C(plan, (hipfftComplex *)d_vout, (hipfftComplex *)d_vout, HIPFFT_BACKWARD);
	abscomplex << <dim3(256, 1, 1), dim3(32, 32, 1) >> >(d_env, d_vout);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mh, start, stop);
	cout << "Gpu Hilbert times = " << mh << "ms\n";

	hipEventRecord(start);
	Gpu_median_filter << <dim3(780, 1, 1), dim3(8, 128, 1) >> >(d_env, d_env, SIGNAL_SIZE, SCAN_LINE); // (x/FREQ_SAMPLING*SOUND_SPEED/2*100) = cm , if 12 cm x=6234 ,6234/8 = 780 Fullpic948
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mf, start, stop);
	cout << "Gpu Median Filter times = " << mf << "ms\n";

	hipEventRecord(start);
	find_maximum << < 32, 256 >> >(d_env, d_max, d_mutex, SIGNAL_SIZE*SCAN_LINE); //<-danger
	logCompressDB << <dim3(256, 1, 1), dim3(32, 32, 1) >> >(d_env, d_max);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ml, start, stop);
	cout << "Gpu LogCompression times = " << ml << "ms\n";

	hipMemcpy(vout, d_env, Imgsize, hipMemcpyDeviceToHost);
	writeFile("D:\\ultrasound\\save.dat", dataLength, vout); //output Vout
	
	for (int i = 0; i < 8192 * 81; i++)
		vout[i] = (vout[i] + 180) / 255;
	Mat A = Mat(81, 8192, CV_64FC1, vout);
	A = A(Rect(0, 0, 6234, 81)); //Crop 6234 = 12  Cm
	resize(A, A, Size(768, 243), CV_INTER_CUBIC);
	transpose(A, A);
	imshow("Image", A);
	waitKey(0);

	delete tdfindex;
	delete raw_data;
	delete max_ps_delay;
	delete elementRxs;
	delete t0;
	delete tdf;
	delete tdds;
	delete tdmin;
	delete tdr;
	delete vout;

	hipfftDestroy(plan);
	hipFree(d_vout);
	hipFree(d_tdr);
	hipFree(d_raw_signal);
	hipFree(d_mutex);
	hipFree(d_max);
	hipFree(d_env);
}
