#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <ctime>

#include "filecon.h"
#include "delaycalc.h"

//Cuda header
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define FREQ_FPGA_CLOCK 100000000 //100Mhz
#define FREQ_SAMPLING	40000000  //40Mhz
#define PITCH			0.0005	  //0.5mn
#define SOUND_SPEED		1540	  //1540m/s
#define SIGNAL_SIZE		8192	  //H
#define CHANNEL			32		  //Point
#define SCAN_LINE		81	      //W
#define NBEFOREPULSE	538
#define NRX				32
#define NTX				32

using namespace std;

int Div0Up(int a, int b)//fix int/int=0
{
	return ((a % b) != 0) ? (1) : (a / b);
}
/*
__device__ void channelcalc(double *sum, int p, int nl, const double *tdr, const double *raw_data)
{
	int Ntdr = tdr[threadIdx.x + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)];
	if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
	{
		*sum += raw_data[Ntdr + (threadIdx.x * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
	}
}
*/

__global__ void beamforming1scanline(int nl, double *vout, const double *tdr, const double *raw_data)
{
	const int nThd = blockDim.x * gridDim.x;
	const int tID = blockIdx.x * blockDim.x + threadIdx.x;
	double sum;
	int Ntdr = 0;
	for (int p = tID; p < SIGNAL_SIZE; p += nThd)//1 scanline 8192 point
	{
		sum = 0;
		//printf("W = %d H = %d\n", nl,threadID);
		for (int i = 0; i < CHANNEL; i++)//1 point = 32 channel
		{
			//channelcalc << <1, 32 >> >(sum, p, nl, tdr, raw_data); // my computer not suppport (compute capability > 3.5)
			Ntdr = tdr[i + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)];
			//printf("%d ", Ntdr);
			if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
			{
				sum += raw_data[Ntdr + (i * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
			}
		}
		vout[p + (nl * SIGNAL_SIZE)] = sum;
		//printf("sum = %lf\n",sum]);
	}
}

void delaysum_beamforming(double *output, const double *tdr, const double *raw_signal)
{
	double sum = 0;
	int Ntdr = 0;
	for (int nl = 0; nl < SCAN_LINE; nl++) //81 scanline
	{
		for (int p = 0; p < SIGNAL_SIZE; p++)//1 scanline 8192 point
		{
			sum = 0;
			for (int i = 0; i < CHANNEL; i++)//1 point = 32 channel
			{
				Ntdr = int( tdr[i + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)] );
				if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
				{
					sum += raw_signal[Ntdr + (i * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
				}
			}
			output[p + (nl * SIGNAL_SIZE)] = sum;
		}
	}
}

int main()
{
	int    dataLength	= SIGNAL_SIZE * SCAN_LINE; // 663552
	int	   Fullsize     = dataLength * CHANNEL *sizeof(double);
	int    Imgsize      = dataLength * sizeof(double);
	//int    ChannelPsize = dataLength * CHANNEL * sizeof(double);
	int	   *tdfindex	= new int   [CHANNEL * SIGNAL_SIZE];
	double *t0			= new double[SCAN_LINE];
	double *max_ps_delay= new double[SCAN_LINE];
	double *tdmin		= new double[SIGNAL_SIZE];
	double *elementRxs	= new double[CHANNEL * SCAN_LINE];
	double *tdf			= new double[2 * CHANNEL * SIGNAL_SIZE];
	double *tdds		= new double[2 * CHANNEL * SIGNAL_SIZE];
	double *vout		= new double[SIGNAL_SIZE * SCAN_LINE];
	double *raw_data	= new double[SIGNAL_SIZE * CHANNEL * SCAN_LINE];
	double *tdr			= new double[SIGNAL_SIZE * CHANNEL * SCAN_LINE];

	double *d_vout;
	double *d_tdr;
	double *d_raw_signal;

	loadRawData("D:\\data.dat", raw_data); // channel*scanline size
	loadData("D:\\loadPsDelay.dat", SCAN_LINE, max_ps_delay);
	loadElementRxs("D:\\loadElementRxs.dat", elementRxs); // channel*scanline size
	for (int i = 0; i < SCAN_LINE; i++) 
		t0[i] = NBEFOREPULSE + (max_ps_delay[i] / FREQ_FPGA_CLOCK * FREQ_SAMPLING); 
	calc_TimeDelay(tdf, tdmin, NTX * 2, PITCH, SOUND_SPEED, FREQ_SAMPLING); // TDF
	calc_tdds(tdds, NRX * 2, tdf, tdmin, FREQ_SAMPLING); //TDDS	
	calc_tdfindex(tdfindex, NRX, elementRxs);// Index TDF
	calc_tdr(tdr, NRX, tdds, tdfindex, t0);//TDR

	//clock_t startTime1 = clock();
	//delaysum_beamforming(vout, tdr, raw_data);
	//cout << "delaysum_beamforming times = "<<double(clock() - startTime1) / (double)CLOCKS_PER_SEC*1000 << " ms." << endl;

	hipMalloc((void **)&d_vout, Imgsize);
	hipMalloc((void **)&d_tdr,Fullsize);
	hipMalloc((void **)&d_raw_signal,Fullsize);

	hipMemcpy(d_tdr, tdr, Fullsize, hipMemcpyHostToDevice);
	hipMemcpy(d_raw_signal, raw_data, Fullsize, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float mi = 0 ,sm = 0;
	
	for (int nl = 0; nl < SCAN_LINE; nl++){
		hipEventRecord(start);
		beamforming1scanline << < 32, 256 >> >(nl, d_vout, d_tdr, d_raw_signal);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&mi, start, stop);
		sm += mi;
	}
	printf("Gpu beamfrom times = %f ms\n",sm);
	hipMemcpy(vout, d_vout, Imgsize, hipMemcpyDeviceToHost);
	writeFile("D:\\save.dat", dataLength, vout); //output Vout

	delete tdfindex;
	delete raw_data;
	delete max_ps_delay;
	delete elementRxs;
	delete t0;
	delete tdf;
	delete tdds;
	delete tdmin;
	delete tdr;
	delete vout;
}
