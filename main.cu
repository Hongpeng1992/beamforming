#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <ctime>

#include "filecon.h"
#include "delaycalc.h"

//Cuda header
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define FREQ_FPGA_CLOCK 100000000 //100Mhz
#define FREQ_SAMPLING	40000000  //40Mhz
#define PITCH			0.0005	  //0.5mn
#define SOUND_SPEED		1540	  //1540m/s
#define SIGNAL_SIZE		8192	  //H
#define CHANNEL			32		  //Point
#define SCAN_LINE		81	      //W
#define NBEFOREPULSE	538
#define NRX				32
#define NTX				32

using namespace std;

int Div0Up(int a, int b)//fix int/int=0
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}
/*
__device__ void channelcalc(double *sum, int p, int nl, const double *tdr, const double *raw_data)
{
	int Ntdr = tdr[threadIdx.x + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)];
	if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
	{
		*sum += raw_data[Ntdr + (threadIdx.x * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
	}
}
*/

__global__ void beamforming1scanline(int nl, float2 *vout, const double *tdr, const double *raw_data)
{
	const int nThd = blockDim.x * gridDim.x;
	const int tID = blockIdx.x * blockDim.x + threadIdx.x;
	double sum;
	int Ntdr = 0;
	for (int p = tID; p < SIGNAL_SIZE; p += nThd)//1 scanline 8192 point
	{
		sum = 0;
		//printf("W = %d H = %d\n", nl,threadID);
		for (int i = 0; i < CHANNEL; i++)//1 point = 32 channel
		{
			//channelcalc << <1, 32 >> >(sum, p, nl, tdr, raw_data); // my computer not suppport (compute capability > 3.5)
			Ntdr = tdr[i + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)];
			//printf("%d ", Ntdr);
			if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
			{
				sum += raw_data[Ntdr + (i * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
			}
		}
		vout[p + (nl * SIGNAL_SIZE)].x = sum;
		//printf("sum = %lf\n",sum]);
	}
}

__global__ void improve(float2 *vout, const double *tdr, const double *raw_data)
{
	const int nThdx = blockDim.x * gridDim.x;
	const int nThdy = blockDim.y * gridDim.y;
	const int tIDx = blockIdx.x * blockDim.x + threadIdx.x;
	const int tIDy = blockIdx.y * blockDim.y + threadIdx.y;
	double sum;
	int Ntdr = 0;
	for (int nl = tIDy; nl < SCAN_LINE; nl += nThdy) //81 scanline
	{
		for (int p = tIDx; p < SIGNAL_SIZE; p += nThdx)//1 scanline 8192 point
		{
			sum = 0;
			//printf("W = %d H = %d\n", nl,threadID);
			for (int i = 0; i < CHANNEL; i++)//1 point = 32 channel
			{
				//channelcalc << <1, 32 >> >(sum, p, nl, tdr, raw_data); // my computer not suppport (compute capability > 3.5)
				Ntdr = tdr[i + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)];
				//printf("%d ", Ntdr);
				if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
				{
					sum += raw_data[Ntdr + (i * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
				}
			}
			vout[p + (nl * SIGNAL_SIZE)].x = sum;
			//printf("sum = %lf\n",sum]);
		}
	}
}

void delaysum_beamforming(double *output, const double *tdr, const double *raw_signal)
{
	double sum = 0;
	int Ntdr = 0;
	for (int nl = 0; nl < SCAN_LINE; nl++) //81 scanline
	{
		for (int p = 0; p < SIGNAL_SIZE; p++)//1 scanline 8192 point
		{
			sum = 0;
			for (int i = 0; i < CHANNEL; i++)//1 point = 32 channel
			{
				Ntdr = int( tdr[i + (p * CHANNEL) + (nl * SIGNAL_SIZE * CHANNEL)] );
				if (Ntdr < SIGNAL_SIZE) //protect out of SIGNAL_SIZE bound
				{
					sum += raw_signal[Ntdr + (i * SIGNAL_SIZE) + (nl * CHANNEL * SIGNAL_SIZE)];//H+C+W
				}
			}
			output[p + (nl * SIGNAL_SIZE)] = sum;
		}
	}
}



int main()
{
	//Cpu mem init
	int    dataLength	= SIGNAL_SIZE * SCAN_LINE; // 663552
	int	   Fullsize     = dataLength * CHANNEL *sizeof(double);
	int    Imgsize      = dataLength * sizeof(double);
	int	   *tdfindex	= new int   [CHANNEL * SIGNAL_SIZE];
	double *t0			= new double[SCAN_LINE];
	double *max_ps_delay= new double[SCAN_LINE];
	double *tdmin		= new double[SIGNAL_SIZE];
	double *elementRxs	= new double[CHANNEL * SCAN_LINE];
	double *tdf			= new double[2 * CHANNEL * SIGNAL_SIZE];
	double *tdds		= new double[2 * CHANNEL * SIGNAL_SIZE];
	double *vout		= new double[SIGNAL_SIZE * SCAN_LINE];
	double *raw_data	= new double[SIGNAL_SIZE * CHANNEL * SCAN_LINE];
	double *tdr			= new double[SIGNAL_SIZE * CHANNEL * SCAN_LINE];
	float2 *vout_com    = new float2[SIGNAL_SIZE * SCAN_LINE];
	//Cuda mem init
	float2 *d_vout;
	double *d_tdr;
	double *d_raw_signal;

	loadRawData("D:\\ultrasound\\loadData.dat", raw_data); // channel*scanline size
	loadData("D:\\ultrasound\\loadPsDelay.dat", SCAN_LINE, max_ps_delay);
	loadElementRxs("D:\\ultrasound\\loadElementRxs.dat", elementRxs); // channel*scanline size
	for (int i = 0; i < SCAN_LINE; i++) 
		t0[i] = NBEFOREPULSE + (max_ps_delay[i] / FREQ_FPGA_CLOCK * FREQ_SAMPLING); 
	calc_TimeDelay(tdf, tdmin, NTX * 2, PITCH, SOUND_SPEED, FREQ_SAMPLING); // TDF
	calc_tdds(tdds, NRX * 2, tdf, tdmin, FREQ_SAMPLING); //TDDS	
	calc_tdfindex(tdfindex, NRX, elementRxs);// Index TDF
	calc_tdr(tdr, NRX, tdds, tdfindex, t0);//TDR

	hipfftHandle plan;
	hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, 1);

	//clock_t startTime1 = clock();
	//delaysum_beamforming(vout, tdr, raw_data);
	//cout << "delaysum_beamforming times = "<<double(clock() - startTime1) / (double)CLOCKS_PER_SEC*1000 << " ms." << endl;

	hipMalloc((void **)&d_vout, SIGNAL_SIZE * SCAN_LINE * sizeof(float2));
	hipMalloc((void **)&d_tdr,Fullsize);
	hipMalloc((void **)&d_raw_signal,Fullsize);

	hipMemcpy(d_tdr, tdr, Fullsize, hipMemcpyHostToDevice);
	hipMemcpy(d_raw_signal, raw_data, Fullsize, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float mi = 0 ,sm = 0;
	
	for (int nl = 0; nl < SCAN_LINE; nl++){
		hipEventRecord(start);
		beamforming1scanline << < 32, 256 >> >(nl, d_vout, d_tdr, d_raw_signal);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&mi, start, stop);
		sm += mi;
	}
	cout <<"Gpubasic Beamfrom times = "<<sm<< "ms\n";
	
	hipEventRecord(start);
	improve<< <dim3(256,1,1),dim3(32,32,1) >> >(d_vout, d_tdr, d_raw_signal);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&mi, start, stop);
	cout <<"GpuImprove beamfrom times = "<<mi<< "ms\n";
	hipMemcpy(vout_com, d_vout, SIGNAL_SIZE * SCAN_LINE * sizeof(float2), hipMemcpyDeviceToHost);
	for (int x = 0; x < 8192 * 81; x++)
		vout[x] = vout_com[x].x;

	writeFile("D:\\ultrasound\\save.dat", dataLength, vout); //output Vout

	delete tdfindex;
	delete raw_data;
	delete max_ps_delay;
	delete elementRxs;
	delete t0;
	delete tdf;
	delete tdds;
	delete tdmin;
	delete tdr;
	delete vout;
}
